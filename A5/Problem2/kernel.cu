#include "hip/hip_runtime.h"
/*Start Header
******************************************************************/
/*!
\file kernel.cu
\author Yong Quanyi Marcus, yong.q, 390005818
\par email: yong.q\@digipen.edu
\date October 18, 2020
\brief
	gpu computing functions
Copyright (C) 2020 DigiPen Institute of Technology.
Reproduction or disclosure of this file or its contents without the
prior written consent of DigiPen Institute of Technology is prohibited.
*/
/* End Header
*******************************************************************/

#include <hip/hip_runtime_api.h>
////////////////////////////////////////////////////////////////////

#define BLOCK_SIZE 32

typedef unsigned int uint;

__global__ void MatrixMulGPUCalc(const double* A, const double* B, double* C, uint rowA, uint colA, uint colB)
{
	__shared__ double ASharedMemory[BLOCK_SIZE*BLOCK_SIZE];
	__shared__ double BSharedMemory[BLOCK_SIZE*BLOCK_SIZE];
	
	uint width = colA;

	uint i = blockIdx.x * blockDim.x + threadIdx.x;
	uint j = blockIdx.y * blockDim.y + threadIdx.y;

	uint x = threadIdx.x;
	uint y = threadIdx.y;

	uint colC = colB;
	//load matrices

	double pValue = 0.0;
	uint endP = ((width - 1) / BLOCK_SIZE) + 1;

	for (uint p = 0; p < endP; ++p)
	{
		uint px = p * BLOCK_SIZE + x;
		uint py = p * BLOCK_SIZE + y;

		ASharedMemory[y * BLOCK_SIZE + x] = (px < colA && j < rowA) ? A[j * colA + px] : 0.0;
		BSharedMemory[y * BLOCK_SIZE + x] = (py < colA && i < colB) ? B[py * colB + i] : 0.0;

		__syncthreads();

		//should be correct
		for (uint k = 0; k < BLOCK_SIZE; ++k)
		{
			pValue += ASharedMemory[y * BLOCK_SIZE + k] * BSharedMemory[k * BLOCK_SIZE + x];
		}
		__syncthreads();
	}


	if (j < rowA && i < colB)
	{
		C[j * colC + i] = pValue;
	}

	
}

extern "C" void MatrixMulGPU(
	const double* inA,
	const double* inB,
	double* out,
	uint rowA, 
	uint colA, 
	uint colB
)
{
	dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 DimGrid2((unsigned int)ceil(((float)colB) / BLOCK_SIZE), (unsigned int)ceil(((float)rowA) / BLOCK_SIZE), 1);
	
	MatrixMulGPUCalc<< < DimGrid2, DimBlock>> >(inA, inB, out, rowA, colA, colB);
	getLastCudaError("MatrixMul failed\n");
	checkCudaErrors(hipDeviceSynchronize());
}


hipStream_t next_stream(hipStream_t* streams, size_t num_streams)
{
	size_t i{}; 
	for(;;) {
		bool operations_pending = hipStreamQuery(streams[i]) == hipErrorNotReady;
		if(!operations_pending)
			break;
		i = (i + 1)%num_streams;
	}

	return streams[i];
}

struct mat_mul
{
	void host_pin_memory();
	void partial_calc();
	void copy_back();
	
	// client data
	const double* inA;
	const double* inB;
	double* out;
	uint rowA;
	uint colA;
	uint colB;
	uint m; 
	uint n;

	// streams
	hipStream_t* streams;
	size_t s;

	// host pinned memory
	double* h_A;
	double* h_B;
	double* h_C;

	//device memory
	double* d_A;
	double* d_B;
	double* d_C;

	//events 
	hipEvent_t host_memory_avail, host_memory_ready;
	
};


void mat_mul::host_pin_memory(void)
{
	uint rowB = colA;
	// loop through all column tiles for A
	for(uint i = 0; i < colA; ++i) {
		//loop over row tiles per A column tile
		for(uint j = 0; j < rowA; ++j) {
			//loop over column tile per B row tile
			for(uint k = 0; k < colB; ++k) {

				// get next stream
				auto stream = next_stream(streams, s);
				//just to be safe
				hipStreamSynchronize(stream);
				
				//wait for host to be available
				hipEventSynchronize(host_memory_avail);
				
				//hipStreamWaitEvent(stream, event);
				hipEventRecord(host_memory_ready);

			}
		}
	}
}

void mat_mul::partial_calc(void)
{
	uint rowB = colA;	
	// loop through all column tiles for A
	for(uint i = 0; i < colA; ++i) {
		//loop over row tiles per A column tile
		for(uint j = 0; j < rowA; ++j) {
			//loop over column tile per B row tile
			for(uint k = 0; k < colB; ++k) {
				

				
				
				hipEventSynchronize(host_memory_ready);
				
				// checkCudaErrors(hipMemcpyAsync((void **) &d_A, m * sizeof(double)));
				// checkCudaErrors(hipMalloc((void **) &d_B, n * sizeof(double)));
				// // output device memory
				// checkCudaErrors(hipMalloc((void **) &d_C, m * n * sizeof(double)));
				//do memcpy
				hipEventRecord(host_memory_avail);
			}
		}
	}
}

void mat_mul::copy_back(void)
{
	uint rowB = colA;
	// loop through all column tiles for A
	for(uint i = 0; i < colA; ++i) {
		//loop over row tiles per A column tile
		for(uint j = 0; j < rowA; ++j) {
			//loop over column tile per B row tile
			for(uint k = 0; k < colB; ++k) {




			}
		}
	}
}

//necessary includes
#include <thread>
#include <array>
#include <vector>

static constexpr size_t num_threads_ = 3;

extern "C" void MatrixMulGPUStream(
	const double* inA,
	const double* inB,
	double* out,
	uint rowA,
	uint colA,
	uint colB,
	uint m,
	uint n,
	uint s
)
{
	// initializing... 
	// allocate pinned memory
	double* h_A;
	double* h_B;
	double* h_C;

	// input host memory
	checkCudaErrors(hipHostAlloc((void **) &h_A, m * sizeof(double), hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc((void **) &h_B, n * sizeof(double), hipHostMallocMapped));
	// output host memory
	checkCudaErrors(hipHostAlloc((void **) &h_C, m * n * sizeof(double), hipHostMallocMapped));
	
	
	double* d_A;
	double* d_B;
	double* d_C;
	// input device memory
	checkCudaErrors(hipMalloc((void **) &d_A, m * sizeof(double)));
	checkCudaErrors(hipMalloc((void **) &d_B, n * sizeof(double)));
	// output device memory
	checkCudaErrors(hipMalloc((void **) &d_C, m * n * sizeof(double)));

	hipEvent_t host_memory_avail, host_memory_ready;
	checkCudaErrors(hipEventCreate(&host_memory_avail));
	checkCudaErrors(hipEventCreate(&host_memory_ready));

	std::vector<hipStream_t> streams{(size_t)s, nullptr};
	for(auto& stream : streams) {
		checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	}

	mat_mul App{};

	//client data
	App.inA = inA;
	App.inB = inB;
	App.out = out;
	App.rowA = rowA;
	App.colA = colA;
	App.colB = colB;
	App.m = m;
	App.n = n;

	// streams
	App.streams = streams.data();
	App.s = streams.size();

	// host pinned memory 
	App.h_A = h_A;
	App.h_B = h_B;
	App.h_C = h_C;

	//device memory
	App.d_A = d_A;
	App.d_B = d_B;
	App.d_C = d_C;

	//events 
	App.host_memory_avail = host_memory_avail;
	App.host_memory_ready = host_memory_ready;


	//start running all here ...
	auto t1 = [&App]() -> void { 
		App.host_pin_memory();
	};
	auto t2 = [&App]() -> void { 
		App.partial_calc();
	};
	auto t3 = [&App]() -> void { 
		App.copy_back();
	};

	std::array<std::thread, num_threads_> threads {
		std::thread{t1}, std::thread{t2}, std::thread{t3}
	};

	for(auto& t: threads) {
		t.join();
	}


	checkCudaErrors(hipHostFree(h_A));
	checkCudaErrors(hipHostFree(h_B));
	checkCudaErrors(hipHostFree(h_C));

	checkCudaErrors(hipFree(d_A));
	checkCudaErrors(hipFree(d_B));
	checkCudaErrors(hipFree(d_C));

	for(auto& stream : streams) {
		checkCudaErrors(hipStreamDestroy(stream));
	}

	checkCudaErrors(hipEventDestroy(host_memory_avail));
	checkCudaErrors(hipEventDestroy(host_memory_ready));
	
}