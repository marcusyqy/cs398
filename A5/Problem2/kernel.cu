#include "hip/hip_runtime.h"
/*Start Header
******************************************************************/
/*!
\file kernel.cu
\author Yong Quanyi Marcus, yong.q, 390005818
\par email: yong.q\@digipen.edu
\date October 18, 2020
\brief
	gpu computing functions
Copyright (C) 2020 DigiPen Institute of Technology.
Reproduction or disclosure of this file or its contents without the
prior written consent of DigiPen Institute of Technology is prohibited.
*/
/* End Header
*******************************************************************/

#include <hip/hip_runtime_api.h>
////////////////////////////////////////////////////////////////////

#define BLOCK_SIZE 32

typedef unsigned int uint;

__global__ void MatrixMulGPUCalc(const double* A, const double* B, double* C, uint rowA, uint colA, uint colB)
{
	__shared__ double ASharedMemory[BLOCK_SIZE*BLOCK_SIZE];
	__shared__ double BSharedMemory[BLOCK_SIZE*BLOCK_SIZE];
	
	uint width = colA;

	uint i = blockIdx.x * blockDim.x + threadIdx.x;
	uint j = blockIdx.y * blockDim.y + threadIdx.y;

	uint x = threadIdx.x;
	uint y = threadIdx.y;

	uint colC = colB;
	//load matrices

	double pValue = 0.0;
	uint endP = ((width - 1) / BLOCK_SIZE) + 1;

	for (uint p = 0; p < endP; ++p)
	{
		uint px = p * BLOCK_SIZE + x;
		uint py = p * BLOCK_SIZE + y;

		ASharedMemory[y * BLOCK_SIZE + x] = (px < colA && j < rowA) ? A[j * colA + px] : 0.0;
		BSharedMemory[y * BLOCK_SIZE + x] = (py < colA && i < colB) ? B[py * colB + i] : 0.0;

		__syncthreads();

		//should be correct
		for (uint k = 0; k < BLOCK_SIZE; ++k)
		{
			pValue += ASharedMemory[y * BLOCK_SIZE + k] * BSharedMemory[k * BLOCK_SIZE + x];
		}
		__syncthreads();
	}


	if (j < rowA && i < colB)
	{
		C[j * colC + i] = pValue;
	}

	
}

extern "C" void MatrixMulGPU(
	const double* inA,
	const double* inB,
	double* out,
	uint rowA, 
	uint colA, 
	uint colB
)
{
	dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 DimGrid2((unsigned int)ceil(((float)colB) / BLOCK_SIZE), (unsigned int)ceil(((float)rowA) / BLOCK_SIZE), 1);
	
	MatrixMulGPUCalc<< < DimGrid2, DimBlock>> >(inA, inB, out, rowA, colA, colB);
	getLastCudaError("MatrixMul failed\n");
	checkCudaErrors(hipDeviceSynchronize());
}


void host_pin_memory(
	const double* inA,
	const double* inB,
	double* out,
	uint rowA,
	uint colA,
	uint colB
)
{
	uint rowB = colA;
	// loop through all column tiles for A
	for(uint i = 0; i < colA; ++i) {
		//loop over row tiles per A column tile
		for(uint j = 0; j < rowA; ++j) {
			//loop over column tile per B row tile
			for(uint k = 0; k < colB; ++k) {




			}
		}
	}
}

void partial_calc(
	const double* inA,
	const double* inB,
	double* out,
	uint rowA,
	uint colA,
	uint colB
)
{
	uint rowB = colA;	
	// loop through all column tiles for A
	for(uint i = 0; i < colA; ++i) {
		//loop over row tiles per A column tile
		for(uint j = 0; j < rowA; ++j) {
			//loop over column tile per B row tile
			for(uint k = 0; k < colB; ++k) {
				
			}
		}
	}
}

void copy_back(
	const double* inA,
	const double* inB,
	double* out,
	uint rowA,
	uint colA,
	uint colB
)
{
	uint rowB = colA;
	// loop through all column tiles for A
	for(uint i = 0; i < colA; ++i) {
		//loop over row tiles per A column tile
		for(uint j = 0; j < rowA; ++j) {
			//loop over column tile per B row tile
			for(uint k = 0; k < colB; ++k) {




			}
		}
	}
}

//necessary includes
#include <thread>
#include <array>

extern "C" void MatrixMulGPUStream(
	const double* inA,
	const double* inB,
	double* out,
	uint rowA,
	uint colA,
	uint colB
)
{
	// initializing... 

	// allocate pinned memory
	float* h_A;
	float* h_B;
	float* h_C;

	hipHostAlloc((void **) &h_A, N * sizeof(float), hipHostMallocDefault);
	hipHostAlloc((void **) &h_B, N * sizeof(float), hipHostMallocDefault);
	hipHostAlloc((void **) &h_C, N * sizeof(float), hipHostMallocDefault);


	
	//start running all here ...

	auto t1 = [inA, inB, out, rowA, colA, colB]() { 
		host_pin_memory(inA, inB, out, rowA, colA, colB);
	};
	auto t2 = [inA, inB, out, rowA, colA, colB]() { 
		partial_calc(inA, inB, out, rowA, colA, colB);
	};
	auto t3 = [inA, inB, out, rowA, colA, colB]() { 
		copy_back(inA, inB, out, rowA, colA, colB);
	};

	std::array<std::thread, 3> threads {
		std::thread{t1}, std::thread{t2}, std::thread{t3}
	};

	for(auto& t: threads) {
		t.join();
	}
}